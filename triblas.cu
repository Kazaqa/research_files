#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 8
#define N 8
#define lda M
#define ldb N
#define ldc N
#define IDX2C(i,j,ld) (((j)*(ld))+(i))


int main (void){
    hipError_t cudaStatA;
    hipError_t cudaStatB;
    hipError_t cudaStatC;
    hipblasStatus_t stat;
    hipblasHandle_t handle; 

    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT; 
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N; 
    hipblasDiagType_t diag = HIPBLAS_DIAG_NON_UNIT;

    int i, j;
    float* devPtrA;
    float* devPtrB;
    float* devPtrC;
    float* alpha = (float *)1;
    float* a = 0;
    float* b = 0;
    float* c = 0;
    a = (float *)malloc (M * M * sizeof (*a));
    b = (float *)malloc (M * N * sizeof (*b));
    c = (float *)malloc (M * N * sizeof (*c));

    if (!a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }

    for (j = 0; j < M; j++) {
        for (i = 0; i < M; i++) {
            if(i > j){
                a[IDX2C(i,j,M)] = 0;
            }
            else if(rand()%2==0){
                a[IDX2C(i,j,M)] = rand();
            }
            else{
                a[IDX2C(i,j,M)] = rand()*(-1);
            }
        }
    }

    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            if(rand()%2==0){
                b[IDX2C(i,j,M)] = rand();
            }
            else{
                b[IDX2C(i,j,M)] = rand()*(-1);
            }
            c[IDX2C(i,j,M)] = 0;
        }
    }


    cudaStatA = hipMalloc ((void**)&devPtrA, M*M*sizeof(*a));
    cudaStatB = hipMalloc ((void**)&devPtrB, M*N*sizeof(*b));
    cudaStatC = hipMalloc ((void**)&devPtrC, M*N*sizeof(*c));
    if (cudaStatA != hipSuccess) {
        printf ("device memory allocation failed(A)");
        return EXIT_FAILURE;
    }
    if (cudaStatB != hipSuccess) {
        printf ("device memory allocation failed(B)");
        return EXIT_FAILURE;
    }
    if (cudaStatC != hipSuccess) {
        printf ("device memory allocation failed(C)");
        return EXIT_FAILURE;
    }



    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (M, M, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed(A)");
        hipFree (devPtrA);
        hipFree (devPtrB);
        hipFree (devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    stat = hipblasSetMatrix (M, N, sizeof(*b), b, M, devPtrB, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed(B)");
        hipFree (devPtrA);
        hipFree (devPtrB);
        hipFree (devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    stat = hipblasStrmm(handle,side,uplo,trans,diag,M,N,alpha,a,lda,b,ldb,c,ldc);
    
    stat = hipblasGetMatrix (M, N, sizeof(*c), devPtrC, M, c, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrA);
        hipFree (devPtrB);
        hipFree (devPtrC);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    hipFree (devPtrA);
    hipFree (devPtrB);
    hipFree (devPtrC);
    hipblasDestroy(handle);
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            printf ("%7.0f", c[IDX2C(i,j,M)]);
        }
        printf ("\n");
    }
    free(a);
    free(b);
    free(c);
    return EXIT_SUCCESS;
}